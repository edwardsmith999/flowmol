/*
//@HEADER
// ************************************************************************
// 
//   Kokkos: Manycore Performance-Portable Multidimensional Arrays
//              Copyright (2012) Sandia Corporation
// 
// Under the terms of Contract DE-AC04-94AL85000 with Sandia Corporation,
// the U.S. Government retains certain rights in this software.
// 
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are
// met:
//
// 1. Redistributions of source code must retain the above copyright
// notice, this list of conditions and the following disclaimer.
//
// 2. Redistributions in binary form must reproduce the above copyright
// notice, this list of conditions and the following disclaimer in the
// documentation and/or other materials provided with the distribution.
//
// 3. Neither the name of the Corporation nor the names of the
// contributors may be used to endorse or promote products derived from
// this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY SANDIA CORPORATION "AS IS" AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL SANDIA CORPORATION OR THE
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
// LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
// NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
// SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//
// Questions? Contact  H. Carter Edwards (hcedwar@sandia.gov) 
// 
// ************************************************************************
//@HEADER
*/

#include <stdlib.h>
#include <iostream>
#include <sstream>
#include <stdexcept>

#include <Kokkos_Cuda.hpp>
#include <Kokkos_CudaSpace.hpp>

#include <Cuda/Kokkos_Cuda_Internal.hpp>
#include <impl/Kokkos_MemoryTracking.hpp>
#include <impl/Kokkos_Error.hpp>

/*--------------------------------------------------------------------------*/

namespace Kokkos {
namespace Impl {

DeepCopy<HostSpace,CudaSpace>
  ::DeepCopy( void * dst , const void * src , size_t n )
{
  CUDA_SAFE_CALL( hipMemcpy( dst , src , n , hipMemcpyDefault ) );
}

DeepCopy<CudaSpace,HostSpace>
  ::DeepCopy( void * dst , const void * src , size_t n )
{
  CUDA_SAFE_CALL( hipMemcpy( dst , src , n , hipMemcpyDefault ) );
}

DeepCopy<CudaSpace,CudaSpace>
  ::DeepCopy( void * dst , const void * src , size_t n )
{
  CUDA_SAFE_CALL( hipMemcpy( dst , src , n , hipMemcpyDefault ) );
}

} // namespace Impl
} // namespace Kokkos

/*--------------------------------------------------------------------------*/
/*--------------------------------------------------------------------------*/

namespace Kokkos {
namespace {

class CudaMemoryTrackingEntry : public Impl::MemoryTrackingEntry
{
public:

  void * const                    ptr_alloc ;
  const size_t                    size ;
  const size_t                    count ;
  Impl::cuda_texture_object_type  tex_obj ;

  CudaMemoryTrackingEntry( const std::string & arg_label ,
                           const std::type_info & arg_info ,
                           void * const           arg_ptr ,
                           const size_t           arg_size ,
                           const size_t           arg_count )
    : Impl::MemoryTrackingEntry( arg_label , arg_info , arg_ptr , arg_size * arg_count )
    , ptr_alloc( arg_ptr )
    , size( arg_size )
    , count( arg_count )
    , tex_obj( 0 )
    {}

  ~CudaMemoryTrackingEntry();
};

CudaMemoryTrackingEntry::~CudaMemoryTrackingEntry()
{
  std::ostringstream oss;
  bool error = false;
  try {
    Kokkos::Impl::cuda_device_synchronize();
  }
  catch(std::runtime_error & err) {
    error = true;
    oss << err.what() << std::endl;
  }

  if ( tex_obj ) {

  }

  try {
    CUDA_SAFE_CALL( hipFree( ptr_alloc ) );
  }
  catch(std::runtime_error & err) {
    error = true;
    oss << err.what() << std::endl;
  }

  if ( error ) {
    std::cerr << "hipFree( " << ptr_alloc << " ) FAILED for " ;
    Impl::MemoryTrackingEntry::print( std::cerr );
    std::cerr << oss.str() << std::endl;
  }
}

Impl::MemoryTracking & cuda_space_singleton()
{
  static Impl::MemoryTracking self("Kokkos::CudaSpace");
  return self ;
}

bool cuda_space_verify_modifiable( const char * const label )
{
  static const char error_in_parallel[] = "Called with HostSpace::in_parallel()" ;
  static const char error_not_exists[]  = "Called after return from main()" ;

  const char * const error_msg =
    HostSpace::in_parallel() ? error_in_parallel : (
    ! cuda_space_singleton().exists() ? error_not_exists : (const char *) 0 );

  if ( error_msg ) {
    std::cerr << "Kokkos::CudaSpace::" << label << " ERROR : " << error_msg << std::endl ;
  }

  return error_msg == 0  ;
}

}

/*--------------------------------------------------------------------------*/

/*--------------------------------------------------------------------------*/

void * CudaSpace::allocate(
  const std::string    & label ,
  const std::type_info & scalar_type ,
  const size_t           scalar_size ,
  const size_t           scalar_count )
{
  void * ptr = 0 ;

  const size_t size = scalar_size * scalar_count ;

  if ( cuda_space_verify_modifiable("allocate") && size ) {

    try {
      Kokkos::Impl::cuda_device_synchronize();

#if defined( CUDA_VERSION ) && ( 6000 <= CUDA_VERSION ) && defined(KOKKOS_USE_CUDA_UVM)
      CUDA_SAFE_CALL( hipMallocManaged( (void**) &ptr, size, hipMemAttachGlobal) );
#else
      CUDA_SAFE_CALL( hipMalloc( (void**) &ptr, size) );
#endif

      Kokkos::Impl::cuda_device_synchronize();
    }
    catch( std::runtime_error & err ) {
      std::ostringstream msg ;
      msg << "Kokkos::Impl::CudaSpace::allocate( "
          << label
          << " , " << scalar_type.name()
          << " , " << scalar_size
          << " , " << scalar_count
          << " ) FAILED memory allocation\n" 
          << err.what();
      Kokkos::Impl::throw_runtime_exception( msg.str() );
    } 

    cuda_space_singleton().insert(
      new CudaMemoryTrackingEntry( label , scalar_type , ptr , scalar_size , scalar_count ) );
  }

  return ptr ;
}

void CudaSpace::increment( const void * ptr )
{
  if ( cuda_space_verify_modifiable("increment") ) {
    cuda_space_singleton().increment( ptr );
  }
}

void CudaSpace::decrement( const void * ptr )
{
  if ( cuda_space_verify_modifiable("decrement") ) {
    cuda_space_singleton().decrement( ptr );
  }
}

void CudaSpace::print_memory_view( std::ostream & o )
{
  cuda_space_singleton().print( o , std::string("  ") );
}

//----------------------------------------------------------------------------

std::string CudaSpace::query_label( const void * p )
{
  const Impl::MemoryTrackingEntry * entry =
    cuda_space_singleton().query( p );

  return entry ? entry->label : std::string("ERROR NOT FOUND");
}

void CudaSpace::access_error()
{
  const std::string msg("Kokkos::CudaSpace::access_error attempt to execute Cuda function from non-Cuda space" );

  Kokkos::Impl::throw_runtime_exception( msg );
}

void CudaSpace::access_error( const void * const ptr )
{
  std::ostringstream msg ;
  msg << "Kokkos::CudaSpace::access_error:" ;
  msg << " attempt to access Cuda-data labeled(" ;
  msg << query_label( ptr ) ;
  msg << ") from non-Cuda execution" ;
  Kokkos::Impl::throw_runtime_exception( msg.str() );
}

/*--------------------------------------------------------------------------*/

} // namespace Kokkos

#if defined( CUDA_VERSION ) && ( 5000 <= CUDA_VERSION )

namespace Kokkos {
namespace Impl {

::hipTextureObject_t
cuda_texture_object_attach(
  const hipChannelFormatDesc & desc ,
  const void * const            ptr )
{
  if ( 0 == ptr || ! cuda_space_verify_modifiable("texture_object_attach") ) return 0 ;

  const unsigned max_count = 1 << 28 ;

  CudaMemoryTrackingEntry * entry =
    dynamic_cast<CudaMemoryTrackingEntry *>( cuda_space_singleton().query( ptr ) );

  const bool ok_found  = 0 != entry ;
  const bool ok_ptr    = ok_found && ptr == entry->ptr_alloc ;
  const bool ok_count  = ok_found && entry->count < max_count ;

  if ( ok_found && ok_ptr && ok_count ) {

    // Can only create texture object on device architure 3.0 or better

    if ( 0 == entry->tex_obj && 300 <= Cuda::device_arch() ) {

      struct hipResourceDesc resDesc ;
      struct hipTextureDesc  texDesc ;

      memset( & resDesc , 0 , sizeof(resDesc) );
      memset( & texDesc , 0 , sizeof(texDesc) );

      resDesc.resType                = hipResourceTypeLinear ;
      resDesc.res.linear.desc        = desc ;
      resDesc.res.linear.sizeInBytes = entry->size * entry->count ;
      resDesc.res.linear.devPtr      = entry->ptr_alloc ;

      hipCreateTextureObject( & entry->tex_obj, & resDesc, & texDesc, NULL);
    }
  }
  else {
    std::ostringstream msg ;
    msg << "CudaSpace::texture_object_attach( " << ptr << " ) FAILED: " ;

    if ( ! ok_found ) {
      msg << "Not View allocated" ;
    }
    else if ( ! ok_ptr ) {
      msg << "Not the originally allocated View \"" << entry->label << "\"" ;
    }
    else if ( ! ok_count ) {
      msg << "Cuda texture object limit exceeded "
          << max_count << " <= " << entry->count ;
    }
    Kokkos::Impl::throw_runtime_exception( msg.str() );
  }

  return entry->tex_obj ;
}

int cuda_texture_object_release(::hipTextureObject_t tex) {
  return 0;//hipDestroyTextureObject(tex);
}

} // namespace Impl
} // namespace Kokkos

#endif

namespace Kokkos {
namespace Impl {

int cuda_texture_object_release(const void* tex) {
  return 0;
}

} // namespace Impl
} // namespace Kokkos


